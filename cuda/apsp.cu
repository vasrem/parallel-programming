#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>

#define bufSize 700000 


struct timeval startwtime,endwtime;

float *h_a;			// Table at host
float *d_a;			// Table at device
int tsize=0;		// number of rows or columns
size_t size = 0 ;	// size of table( tsize* tsize * sizeof(float*))


void print();
void make_table();
void serial();

__global__ void Kernel1(float *A,int N,int k){

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if ( A[i*N+j] > A[i*N+k] + A[k*N+j] ){
		A[i*N+j] = A[i*N+k] + A[k*N+j];
	}
}

int main(){

	make_table();
	/*print();*/

	gettimeofday(&startwtime,NULL);

	serial();
	
	gettimeofday(&endwtime,NULL);
	printf("Serial time : %lf\n",	(double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6 + endwtime.tv_sec - startwtime.tv_sec));

	// ----------------------------
	//           Kernel 1
	// ----------------------------
	
	make_table();
	gettimeofday(&startwtime,NULL);

	// Alloc device table
	hipMalloc(&d_a,size);

	// Transfer table to device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);	

	// Define dimensions	
	int threads = tsize;	
	dim3 dimBlock(threads,threads);
	dim3 dimGrid(tsize/dimBlock.x,tsize/dimBlock.y);	

	// Do the math
	int k = 0;
	for ( k = 0 ; k < tsize ; k++){
		Kernel1<<<dimGrid,dimBlock>>>(d_a,tsize,k);
		hipDeviceSynchronize();
	}
	
	// Transfer table to host

	hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);

	// Free device and host memory
	
	hipFree(d_a);
	free(h_a);

	gettimeofday(&endwtime,NULL);
	printf("Kernel 1 time : %lf\n",	(double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6 + endwtime.tv_sec - startwtime.tv_sec));

	return 0;
	
}


/*	serial()
 *	-----
 *	Runs serial Floys Floyd-Warshall's Algorithm
 */
void serial(){
	int i , j , k ;

	for ( i = 0 ; i < tsize ; i++ ){
		for( j = 0 ; j < tsize ; j++ ){
			for( k = 0 ; k < tsize ; k++ ){
				if( h_a[i*tsize+j] > h_a[i*tsize+k] + h_a[k*tsize+j] ){
					h_a[i*tsize+j] = h_a[i*tsize+k] + h_a[k*tsize+j];
				}				
			}
		}
	}
	printf("\nDone Serial.\n");
}


/*	make_table()
 *	------------
 *	Gets the input table at table A
 *	input file has at first line size of table 
 *	and at the other lines the content.
 */
void make_table(){
	FILE * fp;
	char buf[bufSize];
	int i = 0 ;
	int j = 0 ;
 	
	fp = fopen("input.txt","r");
	
	// Read size of table
	fgets(buf,sizeof(buf),fp);
	tsize =(int) atof(buf);
	size = tsize*tsize*sizeof(float);	
	// Alloc the table at host
	h_a =(float *) malloc (size);
	
	// Fill the table
	while(fgets(buf,sizeof(buf),fp)!=NULL){
		for(j = 0 ; j < tsize ; j++ ){
			h_a[i*tsize+j]=atof(&buf[16*j]);
		}
		i++;
	}

	fclose(fp);
	printf("\nDone making table.\n");
}



/*	print()
 *	-------
 *	Prints the table A	
 */
void print(){
	int i = 0;
	int j = 0;
	for(i=0;i<tsize;i++){
		for(j=0;j<tsize;j++){
			if(isinf(h_a[i*tsize+j])){
				printf("%f\t\t",h_a[i*tsize+j]);
			}else{
				printf("%f\t",h_a[i*tsize+j]);
			}
		}
		printf("\n-------------------------\n");
	}

}



